#include "../helper.h"
#include "kernels.cuh"

using namespace std;

#define GPU_RUNS    50

template<int B, int Q, int lgH>
void radixSort(uint32_t *d_A, uint32_t *d_B, uint32_t *h_B, size_t N) {
    unsigned long elementsPerBlock = B*Q;
    // Setup execution parameters
    const int blocks = (N + elementsPerBlock - 1) / elementsPerBlock;
    const int H = 1<<lgH;
    const int CHUNK = (H + B - 1) / B;
    const int passes = (sizeof(uint32_t)*8)/lgH;

    printf("CHUNK: %d\n", CHUNK);
    printf("Blocks: %d\n", blocks);
    // Temporary I/O buffers
    // we use d_A - will be overwritten, is this ok??

    // uint32_t *d_ind;
    // hipMalloc((void **) &d_ind, N);
    // hipMemcpy(d_ind, d_A, N, hipMemcpyDeviceToDevice);

    // We use d_B

    // uint32_t *d_out;
    // hipMalloc((void **) &d_out, N);
    // hipMemset(d_out, 0, N);

    // global Historgram buffer
    uint32_t *glbHist;
    hipMalloc((void **) &glbHist, sizeof(uint32_t)*blocks*H);

    uint32_t *hist_h = (uint32_t *)malloc(sizeof(uint32_t)*blocks*H);
    int cnt = 0;


    // Loop over sizeof(elem)/lgH
    for (int i_cpu = 0; i_cpu < passes; i_cpu++) {
        // globla_hist[blocks][H]
        hipMemset(glbHist, 0, sizeof(uint32_t)*blocks*H);
        memset(hist_h, 0, sizeof(uint32_t)*blocks*H);
        histogramKernel<B, Q, lgH, H, CHUNK><<<blocks, B>>>(d_A, glbHist, N, i_cpu);
        // Pseudo - use kernels from assignments
        // transpose_hist()
        // scan_hist()
        // transpose_scan()
        // Second kernel - Does sorting and scattering into global memory
        cnt = 0;
        // Update d_ind = d_out
        hipMemcpy(hist_h, glbHist, sizeof(uint32_t)*blocks*H, hipMemcpyDeviceToHost);
//        printf("Histogram %d\n", i_cpu);
        for (int i = 0; i < blocks; i++) {
//            printf("Block %d\n", i);
            for (int j = 0; j < H; j++) {
                cnt += hist_h[i*blocks + j];
//                printf("%d ", hist_h[i*blocks + j]);
            }
//            printf("\n");
        }
        printf("%s - cnt=%d\n", (cnt==N) ? "VALID" : "INVALID", cnt);
    }
}

template<int B, int Q, int lgH>
void runRadixSort(uint32_t *d_A, uint32_t *d_B, uint32_t *h_B, size_t N) {
    // dry run
    radixSort<B, Q, lgH>(d_A, d_B, h_B, N);
    hipDeviceSynchronize();
    gpuAssert( hipPeekAtLastError() );


    //double elapsed;
    //struct timeval t_start, t_end, t_diff;
    //gettimeofday(&t_start, NULL);

//    for(int i=0; i<GPU_RUNS; i++) {
        // IMPLEMENT RADIX SORT!

  //  }
    //hipDeviceSynchronize();

//    gettimeofday(&t_end, NULL);
//    timeval_subtract(&t_diff, &t_end, &t_start);
//    elapsed = (t_diff.tv_sec*1e6+t_diff.tv_usec) / GPU_RUNS;

    // CHANGE TO MEMORY BOUND PERFORMANCE ANALYSIS
//    double  microsecPerSort = elapsed;
    // double flopsPerMatrixMul = 3.0 * M * K * K * N;
    // double gigaFlops = (flopsPerMatrixMul * 1.0e-3f) / microsecPerSort>

    // gpuAssert( hipPeekAtLastError() );

    // Print and validate :)

}

template<int B, int Q, int lgH>
void runAll(size_t N) {
    srand(2025);

    // Allocate host memory for input and output array
    uint32_t *h_A = (uint32_t*)calloc(N, sizeof(uint32_t));
    uint32_t *h_B = (uint32_t*)calloc(N, sizeof(uint32_t));

    // Initialize input array
    // randomInit<uint32_t>(h_A, N);
    for (uint32_t i = 0; i < N; i++) {
        h_A[i] = (i+1) % 256;
    }

    //printf("Array A:\n");
    //for (int i = 0; i < N; i++) {
    //    printf("%d ", h_A[i]);
   // }
    //printf("\n");

    // Allocate device memory
    uint32_t *d_A;
    uint32_t *d_B;
    hipMalloc((void **) &d_A, sizeof(uint32_t)*N);
    hipMalloc((void **) &d_B, sizeof(uint32_t)*N);

    // Copy host memory to device
    hipMemcpy(d_A, h_A, sizeof(uint32_t)*N, hipMemcpyHostToDevice);

    printf("Size of A: %d\n", N);

    // compute efficient radix sort (validation and timing is done in runRadixSort())
    {
        runRadixSort<B, Q, lgH>(d_A, d_B, h_B, N);
    }

    free(h_A);
    free(h_B);
    hipFree(d_A);
    hipFree(d_B);

}

int main(int argc, char *argv[]) {
    if (argc != 2) {
        printf("Usage: %s size-A\n", argv[0]);
        exit(1);
    }
    const size_t SIZE_A = atoi(argv[1]);

    const int B     = 256; // Thread-block size
    const int Q     = 22;  // Number of elements processed by each thread
    const int lgH   = 8;   // Number of bits processed in each pass of counting sort

    runAll<B, Q, lgH>(SIZE_A);

    return 0;
}
