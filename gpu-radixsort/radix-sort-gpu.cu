#include "hip/hip_runtime.h"
#include "../helper.h"
#include "kernels.cuh"

using namespace std;

// FROM ASSIGNMENT 2
uint32_t nextMul32(uint32_t x) {
    return ((x + 31) / 32) * 32;
}

/**
 * FROM ASSIGNMENT 2 - NUMBER OF BLOCKS FOR SCAN!
 * `N` is the input-array length
 * `B` is the CUDA block size
 * This function attempts to virtualize the computation so
 *   that it spawns at most 1024 CUDA blocks; otherwise an
 *   error is thrown. It should not throw an error for any
 *   B >= 64.
 * The return is the number of blocks, and `CHUNK * (*num_chunks)`
 *   is the number of elements to be processed sequentially by
 *   each thread so that the number of blocks is <= 1024.
 */
template<int CHUNK>
uint32_t getNumBlocks(const uint32_t N, const uint32_t B, uint32_t* num_chunks) {
    const uint32_t max_inp_thds = (N + CHUNK - 1) / CHUNK;
    const uint32_t num_thds0    = min(max_inp_thds, MAX_HWDTH);

    const uint32_t min_elms_all_thds = num_thds0 * CHUNK;
    *num_chunks = max(1, (N + min_elms_all_thds - 1) / min_elms_all_thds);

    const uint32_t seq_chunk = (*num_chunks) * CHUNK;
    const uint32_t num_thds = (N + seq_chunk - 1) / seq_chunk;
    const uint32_t num_blocks = (num_thds + B - 1) / B;

    if(num_blocks <= MAX_BLOCK) {
        return num_blocks;
    } else {
        //printf("Warning: reduce/scan configuration does not allow the maximal concurrency supported by hardware.\n");
        const uint32_t num_blocks = 1024;
        const uint32_t num_thds   = num_blocks * B;
        const uint32_t num_conc_elems = num_thds * CHUNK;
        *num_chunks = (N + num_conc_elems - 1) / num_conc_elems;
        return num_blocks;
    }
}

template<int B, int Q, int lgH>
double radixSort(uint32_t *d_A, uint32_t *d_B, uint32_t *h_B, size_t N) {
    unsigned long elementsPerBlock = B*Q;
    // Setup execution parameters

    // For histogram kernel
    const int blocks = (N + elementsPerBlock - 1) / elementsPerBlock;
    const int H = 1<<lgH;
    const int CHUNK = (H + B - 1) / B;
    const int passes = (sizeof(uint32_t)*8)/lgH;
    const int hist_size = blocks*H;
    const int hist_mem_size = sizeof(uint32_t)*hist_size;

    // For transpose kernel
    int  dimy = (blocks+TILE_SIZE-1) / TILE_SIZE;
    int  dimx = (H+TILE_SIZE-1) / TILE_SIZE;
    dim3 block(TILE_SIZE, TILE_SIZE, 1);
    dim3 grid (dimx, dimy, 1);
    dim3 grid2 (dimy, dimx, 1);

    // For scan kernel
    // COPIED from scaninc() in host_skel.cuh assignment-2
    const uint32_t tp_size = sizeof(uint32_t);
    const uint32_t CHUNK_SCAN = ELEMS_PER_THREAD*4 / tp_size;
    uint32_t num_seq_chunks;
    const uint32_t num_blocks = getNumBlocks<CHUNK_SCAN>(hist_size, B, &num_seq_chunks);
    const size_t   shmem_size = B * tp_size * CHUNK_SCAN;

    //

    // printf("CHUNK: %d\n", CHUNK);
    // printf("Blocks: %d\n", blocks);

    // Meassuring performance should not contain all the memory allocation
    // double elapsed;
    // struct timeval t_start, t_end, t_diff;
    // gettimeofday(&t_start, NULL);

    // you need three buffers for the input/output -> first iter reads from d_A, writes to d_B, then write d_b into d_ind (third iteration)
    uint32_t *tmp_inp;
    uint32_t *tmp_out = d_B;
    hipMalloc((void **) &tmp_inp, sizeof(uint32_t)*N);
    hipMemcpy(tmp_inp, d_A, sizeof(uint32_t)*N, hipMemcpyDeviceToDevice);
    uint32_t *sort_mem_ptr = tmp_inp;

    // global Historgram buffer
    uint32_t *hist;
    uint32_t *hist_tr;
    uint32_t *hist_scan;
    uint32_t *hist_scan_tr;
    hipMalloc((void **) &hist, hist_mem_size);
    hipMalloc((void **) &hist_tr, hist_mem_size);
    hipMalloc((void **) &hist_scan, hist_mem_size);
    hipMalloc((void **) &hist_scan_tr, hist_mem_size);

    uint32_t* d_tmp;
    hipMalloc((void**)&d_tmp, MAX_BLOCK*sizeof(uint32_t));

    // Test runtime - should be measured from here
    double elapsed;
    struct timeval t_start, t_end, t_diff;
    gettimeofday(&t_start, NULL);

    // Loop over sizeof(elem)/lgH
    for (int i_cpu = 0; i_cpu < passes; i_cpu++) {
        // globla_hist[blocks][H]
        histogramKernel<B, Q, lgH, H, CHUNK><<<blocks, B>>>(sort_mem_ptr, hist, N, i_cpu);

        // tanspose
        coalsTransposeKer<uint32_t,TILE_SIZE> <<<grid, block>>>
                        (hist, hist_tr, blocks, H);
        // scan
        {
            redAssocKernel<Add<uint32_t>, CHUNK_SCAN><<< num_blocks, B, shmem_size >>>(d_tmp, hist_tr, hist_size, num_seq_chunks);

            {
                const uint32_t block_size = nextMul32(num_blocks);
                const size_t shmem_size = block_size * sizeof(uint32_t);
                scan1Block<Add<uint32_t>><<< 1, block_size, shmem_size>>>(d_tmp, num_blocks);
            }

            scan3rdKernel<Add<uint32_t>, CHUNK_SCAN><<< num_blocks, B, shmem_size >>>(hist_scan, hist_tr, d_tmp, hist_size, num_seq_chunks);
        }

        // transpose
        coalsTransposeKer<uint32_t,TILE_SIZE> <<<grid2, block>>>
                        (hist_scan, hist_scan_tr, H, blocks);

        // Second kernel - Does sorting and scattering into global memory
        partitionScatterKer<B, Q, lgH><<<blocks, B>>>(sort_mem_ptr, N, hist, hist_scan_tr, tmp_out, i_cpu);

        // Update d_ind = d_out
        // pointer swap - DON'T MEMCPY
        tmp_inp = sort_mem_ptr;
        sort_mem_ptr = tmp_out;
        tmp_out = tmp_inp;
    }

    hipMemcpy(d_B, sort_mem_ptr, sizeof(uint32_t)*N, hipMemcpyDeviceToDevice);

    // To here
    gettimeofday(&t_end, NULL);
    timeval_subtract(&t_diff, &t_end, &t_start);
    elapsed = (t_diff.tv_sec*1e6+t_diff.tv_usec);

    // printf("CUB Sorting for N=%lu runs in: %.2f us, Sorted keys per second: %.2f\n", N, elapsed, (N/(elapsed/1e6)));

    hipFree(hist);
    hipFree(hist_tr);
    hipFree(hist_scan);
    hipFree(hist_scan_tr);

    return elapsed;
}

template<int B, int Q, int lgH>
void runRadixSort(uint32_t *d_A, uint32_t *d_B, uint32_t *h_B, size_t N) {
    double elapsed = 0;
    // dry run
    radixSort<B, Q, lgH>(d_A, d_B, h_B, N);
    hipDeviceSynchronize();
    gpuAssert( hipPeekAtLastError() );

    // printf("CUB Sorting for N=%lu runs in: %.2f us, Sorted keys per second: %.2f\n", N, elapsed, (N/(elapsed/1e6)));

    // uint32_t *arr_inp = (uint32_t *)malloc(sizeof(uint32_t)*N);
    // hipMemcpy(arr_inp, d_A, sizeof(uint32_t)*N, hipMemcpyDeviceToHost);
    
    // printf("input array after sort:\n");
    // for (int i = 0; i < N; i++) {
    //     printf("%d ", arr_inp[i]);
    // }
    // printf("\n\n");

    // double elapsed;
    // struct timeval t_start, t_end, t_diff;
    // gettimeofday(&t_start, NULL);

    // radixSort<B, Q, lgH>(d_A, d_B, h_B, N);

    for(int i=0; i<GPU_RUNS; i++) {
         elapsed += radixSort<B, Q, lgH>(d_A, d_B, h_B, N);
    }
    hipDeviceSynchronize();

    // gettimeofday(&t_end, NULL);
    // timeval_subtract(&t_diff, &t_end, &t_start);
    // elapsed = (t_diff.tv_sec*1e6+t_diff.tv_usec) / ((double)GPU_RUNS);
    // elapsed = elapsed / ((double)GPU_RUNS);

    // // CHECK MEMORY BOUND PERFORMANCE ANALYSIS!
    // // double gigaBytesPerSec = N * sizeof(uint32_t) * 1.0e-3f / elapsed;
    // printf("CUB Sorting for N=%lu runs in: %.2f us, Sorted keys per second: %.2f\n", N, elapsed, (N/(elapsed/1e6)));
    // // printf("Radix sort of uint32_t GPU runs in: %.2f microsecs, GB/sec: %.2f\n"
    // //           , elapsed, gigaBytesPerSec);

    // gpuAssert( hipPeekAtLastError() );
    printf("CUB Sorting for N=%lu runs in: %.2f us, Sorted keys per second: %.2f\n", N, elapsed / ((double)GPU_RUNS), (N/(elapsed/1e6)));

    // // Print and validate :)
    printf("Validating result... ");
    hipMemcpy(h_B, d_B, sizeof(uint32_t)*N, hipMemcpyDeviceToHost);
    validate<uint32_t>(h_B, N);

    // printf("sorted array after sort:\n");
    // for (int i = 0; i < N; i++) {
    //     printf("%d ", h_B[i]);
    // }
    // printf("\n");

    // printf("Sorted array:\n");
    // for (int i = 0; i < 3000; i++) {
    //     printf("%d ", h_B[i]);
    // }
    // printf("\n\n");
    // for (int i = N-3000; i < N; i++) {
    //     printf("%d ", h_B[i]);
    // }
    // printf("\n\n");

}

template<int B, int Q, int lgH>
void runAll(size_t N) {
    srand(2025);

    // Allocate host memory for input and output array
    uint32_t *h_A = (uint32_t*)calloc(N, sizeof(uint32_t));
    uint32_t *h_B = (uint32_t*)calloc(N, sizeof(uint32_t));

    // Initialize input array
    randomInit<uint32_t>(h_A, N);
    // for (uint32_t i = 0; i < N; i++) {
    //     h_A[i] = i % 256;
    // }

    // printf("Array A:\n");
    // for (int i = 0; i < N; i++) {
    //    printf("%d ", h_A[i]);
    // }
    // printf("\n");

    // Allocate device memory
    uint32_t *d_A;
    uint32_t *d_B;
    hipMalloc((void **) &d_A, sizeof(uint32_t)*N);
    hipMalloc((void **) &d_B, sizeof(uint32_t)*N);

    // Copy host memory to device
    hipMemcpy(d_A, h_A, sizeof(uint32_t)*N, hipMemcpyHostToDevice);

    printf("Size of A: %d\n", N);

    // compute efficient radix sort (validation and timing is done in runRadixSort())
    {
        runRadixSort<B, Q, lgH>(d_A, d_B, h_B, N);
    }

    free(h_A);
    free(h_B);
    hipFree(d_A);
    hipFree(d_B);

}

int main(int argc, char *argv[]) {
    if (argc != 2) {
        printf("Usage: %s size-A\n", argv[0]);
        exit(1);
    }

    hipSetDevice(1);
    initHwd();

    const size_t SIZE_A = atoi(argv[1]);

    const int B     = 256; // Thread-block size
    const int Q     = 22;  // Number of elements processed by each thread
    const int lgH   = 8;   // Number of bits processed in each pass of counting sort

    runAll<B, Q, lgH>(SIZE_A);

    return 0;
}
